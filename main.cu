#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>

#include "Vec3.h"
#include "Ray.h"
#include "Hitable.h"
#include "Sphere.h"
#include "HitableList.h"
#include "Image.h"
#include "Camera.h"
#include "Material.h"



// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
              file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}


// final color function
__device__ Vec3 color(const Ray& r, Hitable **world, hiprandState *local_rand_state) {
  Ray cur_ray = r;
  Vec3 cur_attenuation = Vec3(1.0,1.0,1.0);
  for(int i = 0; i < 50; i++) {
    HitRecord rec;
    if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
      Ray scattered;
      Vec3 attenuation;
      if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
        cur_attenuation *= attenuation;
        cur_ray = scattered;
      }
      else {
        return {0.0,0.0,0.0};
      }
    }
    else {
      Vec3 unit_direction = unit_vector(cur_ray.direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);
      Vec3 c = (1.0f-t)*Vec3(1.0, 1.0, 1.0) + t*Vec3(0.5, 0.7, 1.0);
      return cur_attenuation * c;
    }
  }
  return {0.0,0.0,0.0}; // exceeded recursion
}

__global__ void render_init(int nx, int ny, hiprandState *rand_state, unsigned long long SEED) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= nx) || (j >= ny)) return;
  int pixel_index = j*nx + i;
  hiprand_init(SEED, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3 *fb, int max_x, int max_y, int ns,
                       Camera ** cam, Hitable **world, hiprandState *rand_state){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j*max_x + i;
  hiprandState local_rand_state = rand_state[pixel_index];
  Vec3 px_color(0,0,0);
  for(int s=0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
    Ray r = (*cam)->getRay(u, v, &local_rand_state);
    px_color += color(r, world, &local_rand_state);
  }
  rand_state[pixel_index] = local_rand_state;
  px_color /= float(ns);
  px_color[0] = sqrt(px_color[0]);
  px_color[1] = sqrt(px_color[1]);
  px_color[2] = sqrt(px_color[2]);
  fb[pixel_index] = px_color;
}

__global__ void create_world(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N) {

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5, new lambertian(Vec3(0.8, 0.2, 0.3)));
    *(d_list+1) = new Sphere(Vec3(0, -100.5, -1), 100, new lambertian(Vec3(0.8, 0.8, 0.1)));
    *(d_list+2) = new Sphere(Vec3(1, 0, -1), 0.5, new metal(Vec3(0.8, 0.6, 0.2), 0.0));
    *(d_list+3) = new Sphere(Vec3(-1, 0, -1), 0.5, new dielectric(1.5));
    *(d_list+4) = new Sphere(Vec3(-1, 0, -1), -0.45, new dielectric(1.5));
    *d_world    = new HitableList(d_list, object_N);
    Vec3 lookfrom(3,3,2);
    Vec3 lookat(0,0,-1);
    float dist_to_focus = (lookfrom-lookat).length();
    float aperture = 2.0;
    *d_camera   = new Camera(lookfrom,
                             lookat,
                             Vec3(0,1,0),
                             20.0,
                             float(nx)/float(ny),
                             aperture,
                             dist_to_focus);
  }
}

__global__ void free_world(Hitable **d_list, Hitable **d_world, Camera **d_camera) {
  for(int i=0; i < 5; i++) {
    delete ((Sphere*)d_list[i])->mat_ptr; // this line makes the code crash ! ! ! ! ! ! ! !
    delete d_list[i];
  }

  delete *d_world;
  delete *d_camera;

}


int main() {
  const int nx = 800;
  const int ny = 400;
  const int ns = 100;
  int tx = 8;
  int ty = 8;
  const int rSEED = 1;

  int num_pixels = nx*ny;
  size_t fb_size = 3 * num_pixels * sizeof(float);

  const int object_N = 5;

  Hitable **d_list;
  checkCudaErrors(   hipMalloc(  (void **)&d_list  , object_N*sizeof(Hitable *)));
  Hitable **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hitable *)));
  Camera **d_camera;
  checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));

  create_world<<<1,1>>>(d_list,d_world, d_camera, nx, ny, object_N);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

//  free_world<<<1,1>>>(d_list,d_world, d_camera);
//  checkCudaErrors(hipGetLastError());
//  checkCudaErrors(hipDeviceSynchronize());

  // allocate FB
  Vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size)   );

  // allocate a cuRAND d_rand_state object for every pixel
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));



  dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
  dim3 threads(tx,ty);

  //initialize RNG
  render_init<<<blocks, threads>>>(nx, ny, d_rand_state, rSEED);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  auto start = std::chrono::high_resolution_clock::now();
  // main render function
  render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

  // make image to write to
  Image image(nx, ny);
  // Generate the image
  for (int j = ny - 1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      image.write_pixel(i, ny - 1 - j, fb[j*nx+i]);
    }
  }

  image.save("../output.png");

  checkCudaErrors(hipDeviceSynchronize());
  free_world<<<1,1>>>(d_list,d_world, d_camera);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_camera));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_rand_state));
  checkCudaErrors(hipFree(fb));

  hipDeviceReset();

  return 0;
}