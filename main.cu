#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_texture_types.h"
#include "hip/hip_runtime.h"

#include "Vec3.h"
#include "Ray.h"
#include "Hitable.h"
#include "Sphere.h"
#include "HitableList.h"
#include "Image.h"
#include "Camera.h"
#include "Material.h"
#include "Texture.h"

#include <filesystem>



// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
              file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}


// final color function
__device__ Vec3 color(const Ray& r, Hitable **world, hiprandState *local_rand_state) {
  Ray cur_ray = r;
  Vec3 cur_attenuation = Vec3(1.0,1.0,1.0);
  for(int i = 0; i < 50; i++) {
    HitRecord rec;
    if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
      Ray scattered;
      Vec3 attenuation;
      if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
        cur_attenuation *= attenuation;
        cur_ray = scattered;
      }
      else {
        return {0.0,0.0,0.0};
      }
    }
    else {
      Vec3 unit_direction = unit_vector(cur_ray.direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);
      Vec3 c = (1.0f-t)*Vec3(1.0, 1.0, 1.0) + t*Vec3(0.5, 0.7, 1.0);
      return cur_attenuation * c;
    }
  }
  return {0.0,0.0,0.0}; // exceeded recursion
}

__global__ void render_init(int nx, int ny, hiprandState *rand_state, unsigned long long SEED) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= nx) || (j >= ny)) return;
  int pixel_index = j*nx + i;
  hiprand_init(SEED, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3 *fb, int max_x, int max_y, int ns,
                       Camera ** cam, Hitable **world, hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j*max_x + i;
  hiprandState local_rand_state = rand_state[pixel_index];
  Vec3 px_color(0,0,0);
  for(int s=0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
    Ray r = (*cam)->getRay(u, v, &local_rand_state);
    px_color += color(r, world, &local_rand_state);
  }
  rand_state[pixel_index] = local_rand_state;
  px_color /= float(ns);
  px_color[0] = sqrt(px_color[0]);
  px_color[1] = sqrt(px_color[1]);
  px_color[2] = sqrt(px_color[2]);
  fb[pixel_index] = px_color;
}

__global__ void create_world(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N, hipTextureObject_t textureObject) {

  if (threadIdx.x == 0 && blockIdx.x == 0) {
//       from the d_img_data, create the ImageTexture object
    ImageTexture* earth_img = new ImageTexture(textureObject);

    Texture *bigSphereChecker = new CheckerTexture(new ConstantTexture(Vec3(0.2, 0.3, 0.1)), new ConstantTexture(Vec3(0.9, 0.9, 0.9)));

    *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5, new lambertian(earth_img));
//    *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5, new lambertian(new ConstantTexture(Vec3(0.8, 0.2, 0.3))));
    *(d_list+1) = new Sphere(Vec3(0, -100.5, -1), 100, new lambertian(bigSphereChecker));
    *(d_list+2) = new Sphere(Vec3(1, 0, -1), 0.5, new metal(new ConstantTexture(Vec3(0.8, 0.6, 0.2)), 0.0f));
    *(d_list+3) = new Sphere(Vec3(-1, 0, -1), 0.5, new dielectric(1.5));
    *(d_list+4) = new Sphere(Vec3(-1, 0, -1), -0.45, new dielectric(1.5));
    *d_world    = new HitableList(d_list, object_N);
    Vec3 lookfrom(3,3,2);
    Vec3 lookat(0,0,-1);
    float dist_to_focus = (lookfrom-lookat).length();
    float aperture = 0.3;
    *d_camera   = new Camera(lookfrom,
                             lookat,
                             Vec3(0,1,0),
                             40.0,
                             float(nx)/float(ny),
                             aperture,
                             dist_to_focus, 0.0f, 1.0f);
  }
}

__global__ void free_world(Hitable **d_list, Hitable **d_world, Camera **d_camera) {
  for(int i=0; i < 5; i++) {
    delete ((Sphere*)d_list[i])->mat_ptr;
    delete d_list[i];
  }

  delete *d_world;
  delete *d_camera;

}



__global__ void debug_texture_kernel(hipTextureObject_t tex, float* output, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    float u = (float)x / (float)width;
    float v = (float)y / (float)height;
    float4 texel = tex2D<float4>(tex, u, 1.0f - v);
    int idx = (y * width + x) * 3;
    output[idx] = texel.x;
    output[idx+1] = texel.y;
    output[idx+2] = texel.z;
  }
}


hipTextureObject_t createImageTexture(const char *const filename){

//  /// texture allocation begin

  int width, height, channels;
  unsigned char* img = stbi_load(filename, &width, &height, &channels, 0);
  if (!img) {
    // Handle error
    fprintf(stderr, "Failed to load image\n");
    exit(2);
  }
  printf("width: %d, height: %d, channels: %d\n", width, height, channels);

  // allocate texture array on device and copy image data to it

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  hipArray* cuArray;
  hipMallocArray(&cuArray, &channelDesc, width, height);

  hipMemcpy2DToArray(cuArray, 0, 0, img, width * channels, width * channels, height, hipMemcpyHostToDevice);

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeNormalizedFloat;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  stbi_image_free(img);

  return texObj;
}




int main() {
  const int nx = 800;
  const int ny = 400;
  const int ns = 1000;
  int tx = 8;
  int ty = 8;
  const int rSEED = 1;

  int num_pixels = nx*ny;
  size_t fb_size = 3 * num_pixels * sizeof(float);

  const int object_N = 5;

  // load texture into GPU memory
  hipTextureObject_t texObj = createImageTexture("../earthmap1kpng.png");

  /// texture allocation end

  Hitable **d_list;
  checkCudaErrors(   hipMalloc(  (void **)&d_list, object_N*sizeof(Hitable *)));
  Hitable **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hitable *)));
  Camera **d_camera;
  checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));

  create_world<<<1,1>>>(d_list,d_world, d_camera, nx, ny, object_N, texObj);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // allocate FB
  Vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size)   );

  // allocate a cuRAND d_rand_state object for every pixel
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));


  dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
  dim3 threads(tx,ty);

  //initialize RNG
  render_init<<<blocks, threads>>>(nx, ny, d_rand_state, rSEED);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  auto start = std::chrono::high_resolution_clock::now();
  // main render function
  render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

  // make image to write to
  Image image(nx, ny);
  // Generate the image
  for (int j = ny - 1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      image.write_pixel(i, ny - 1 - j, fb[j*nx+i]);
    }
  }

//  stbi_image_free(img);
  image.save("../output.png");

  checkCudaErrors(hipDeviceSynchronize());
  free_world<<<1,1>>>(d_list,d_world, d_camera);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_camera));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_rand_state));
  checkCudaErrors(hipFree(fb));

  hipDeviceReset();

  return 0;
}