#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_texture_types.h"
#include "hip/hip_runtime.h"

#include "Vec3.h"
#include "Ray.h"
#include "hitable/Hitable.h"
#include "hitable/Sphere.h"
#include "hitable/HitableList.h"
#include "util/Image.h"
#include "Camera.h"
#include "Material.h"
#include "Texture.h"
#include "ConfigParser.h"
#include "AA_Rectangles.h"
#include "Box.h"
#include "hitable/Triangle.h"
#include "hitable/Mesh.h"
#include "util/ObjFile.h"

#include <filesystem>
#include <fstream>



// final color function
__device__ Vec3 color(const Ray& r, Hitable **world, hiprandState *local_rand_state) {
  Ray cur_ray = r;
  Vec3 cur_attenuation = {1.0,1.0,1.0};
  Vec3 result = {0,0,0};

  for(int i = 0; i < 50; i++) {
    HitRecord rec;

    // does the ray hit anything?
    if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
      Vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);

      Ray scattered;
      Vec3 attenuation;

      if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
        result += cur_attenuation * emitted;
        cur_attenuation *= attenuation;
        cur_ray = scattered;
      }
      else {
        result += cur_attenuation * emitted;

        break;
      }
    }
    else {
      // the ray did not his anything, return background color times current attenuation..!
//      Vec3 unit_direction = unit_vector(cur_ray.direction());
//      float t = 0.5f*(unit_direction.y() + 1.0f);
//      Vec3 background = (1.0f-t)*Vec3(0.5, 0.5, 0.75) + t*Vec3(0.3, 0.5, 0.7);

      // c is background color
      Vec3 background = Vec3(0.0, 0.0, 0.0);
      result += cur_attenuation * background; // maybe * instead? or just =
      break;
    }
  }
  return result; // exceeded recursion
}

__global__ void render_init(int nx, int ny, hiprandState *rand_state, unsigned long long SEED) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= nx) || (j >= ny)) return;
  int pixel_index = j*nx + i;
  hiprand_init(SEED, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3 *fb, int max_x, int max_y, int ns,
                       Camera ** cam, Hitable **world, hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j*max_x + i;
  hiprandState local_rand_state = rand_state[pixel_index];
  Vec3 px_color(0,0,0);
  for(int s=0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
    Ray r = (*cam)->getRay(u, v, &local_rand_state);
    px_color += color(r, world, &local_rand_state);
  }
  rand_state[pixel_index] = local_rand_state;
  px_color /= float(ns);
  px_color[0] = sqrt(px_color[0]);
  px_color[1] = sqrt(px_color[1]);
  px_color[2] = sqrt(px_color[2]);
  fb[pixel_index] = px_color;
}

//__global__ void make_camera(const int nx, const int ny, Camera **d_camera,
//                            const Vec3& lookfrom, const Vec3& lookat, const Vec3& vup,
//                            const float vfov, const float aperture, const float focus_dist){
////  Vec3 lookfrom(3,3,2);
////  Vec3 lookat(0,0,-1);
////  float focus_dist = (lookfrom-lookat).length();
////  float aperture = 0.3;
//  *d_camera   = new Camera(lookfrom,
//                           lookat,
//                           Vec3(0,1,0),
//                           vfov,
//                           float(nx)/float(ny),
//                           aperture,
//                           focus_dist, 0.0f, 1.0f);
//}

__global__ void create_spheres_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N, hipTextureObject_t textureObject) {

  if (threadIdx.x == 0 && blockIdx.x == 0) {
//  create ImageTexture from cuda texture object
    ImageTexture* earth_img = new ImageTexture(textureObject);

    Texture *bigSphereChecker = new CheckerTexture(new ConstantTexture(Vec3(0.2, 0.3, 0.1)), new ConstantTexture(Vec3(0.9, 0.9, 0.9)));

    *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5, new lambertian(earth_img));
//    *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5, new lambertian(new ConstantTexture(Vec3(0.8, 0.2, 0.3))));
    *(d_list+1) = new Sphere(Vec3(0, -100.5, -1), 100, new lambertian(bigSphereChecker));
    *(d_list+2) = new Sphere(Vec3(1, 0, -1), 0.5, new DiffuseLight(new ConstantTexture(Vec3(1,0.9,(float)135/255))));
//    *(d_list+2) = new Sphere(Vec3(1, 0, -1), 0.5, new metal(new ConstantTexture(Vec3(0.8, 0.6, 0.2)), 0.0f));
    *(d_list+3) = new Sphere(Vec3(-1, 0, -1), 0.5, new dielectric(1.5));
    *(d_list+4) = new Sphere(Vec3(-1, 0, -1), -0.45, new dielectric(1.5));
    //triangle test
    *(d_list+5) = new Triangle(Vec3(0, 2, -2), Vec3(2, -1.2, -2), Vec3(-2, -1.2, -2), new lambertian(new ConstantTexture(Vec3(0.8, 0.2, 0.3))));
    *d_world    = new HitableList(d_list, object_N);



    Vec3 lookfrom(3,3,2);
    Vec3 lookat(0,0,-1);
    float dist_to_focus = (lookfrom-lookat).length();
    float aperture = 0.3;
    *d_camera   = new Camera(lookfrom,
                             lookat,
                             Vec3(0,1,0),
                             40.0,
                             float(nx)/float(ny),
                             aperture,
                             dist_to_focus, 0.0f, 1.0f);
  }
}

__global__ void free_spheres_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera) {
  for(int i=0; i < 6; i++) {
//    delete ((Sphere*)d_list[i])->mat_ptr;
    delete d_list[i];
  }

  delete *d_world;
  delete *d_camera;
}

__global__ void create_cornell_box_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    Material *red = new lambertian(new ConstantTexture(Vec3(0.65, 0.05, 0.05)));
    Material *white = new lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73)));
    Material *green = new lambertian(new ConstantTexture(Vec3(0.12, 0.45, 0.15)));
    Material *light = new DiffuseLight(new ConstantTexture(Vec3(15, 15, 15)));
    Material *shiny = new metal(new ConstantTexture(Vec3(1.0, 1.0, 1.0)), 0.03f);

    *(d_list) = new flip_normals(new YZ_Rectangle(0, 555, 0, 555, 555, green));
    *(d_list+1) = new YZ_Rectangle(0, 555, 0, 555, 0, red);
    *(d_list+2) = new XZ_Rectangle(213, 343, 227, 332, 554, light);
    *(d_list+3) = new flip_normals( new XZ_Rectangle(0, 555, 0, 555, 555, white));
    *(d_list+4) = new XZ_Rectangle(0, 555, 0, 555, 0, white);
    *(d_list+5) = new flip_normals( new XY_Rectangle(0, 555, 0, 555, 555, white));



    *(d_list+6) = new translate( new rotate_y( new Box(Vec3(0, 0, 0), Vec3(165, 165, 165), new dielectric(1.5)), -18), Vec3(130, 0, 65));
    *(d_list+7) = new translate( new rotate_y( new Box(Vec3(0, 0, 0), Vec3(165, 330, 165), shiny), 15), Vec3(265, 0, 295));



    *d_world = new HitableList(d_list, object_N);
    Vec3 lookfrom(278, 278, -800);
    Vec3 lookat(278, 278, 0);
    float dist_to_focus = 10.0;
    float aperture = 0.0;
    *d_camera = new Camera(lookfrom,
                           lookat,
                           Vec3(0, 1, 0), 40.0,
                           float(nx) / float(ny),
                           aperture,
                           dist_to_focus, 0.0f, 1.0f);
  }
}

__global__ void free_cornell_box_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera) {

  //memory leak here for now but using virtual destructors causes crash for some reason..
  for(int i=0; i < 8; i++) {
    delete d_list[i];
  }

  delete *d_world;
  delete *d_camera;
}

__global__ void meshFromTriangleArray(Hitable **d_mesh, Hitable **d_MeshTriangles, Vec3 *d_points, size_t nPoints, Vec3 *d_faces, size_t nFaces) {
  int l = 0;
  Material *mat = new lambertian(new ConstantTexture(Vec3(0.8, 0.2, 0.3)));
  for (int i = 0; i < nFaces; i++) {
    Vec3 face = d_faces[i];
    Vec3 p0 = d_points[(int)face[0]]; // get the points from the array
    Vec3 p1 = d_points[(int)face[1]];
    Vec3 p2 = d_points[(int)face[2]];
    *(d_MeshTriangles + l) = new Triangle(p0, p1, p2, mat); // fill up the array with Triangle hitables
    l++;
  }

  *d_mesh = new HitableList(d_MeshTriangles, l); // now create the HitableList with triangle array
}


__host__ Hitable** loadMeshFromOBJFile(const std::string &filename, float scale = 1.0f) {

  ObjFile teapot_obj(filename);

  size_t nPoints = teapot_obj.points.size();
  size_t pointsArraySize = nPoints * sizeof(Vec3);
  size_t nFaces = teapot_obj.faces.size();
  size_t facesArraySize = nFaces * sizeof(Vec3);

  // scale the mesh
  for (int i = 0; i < nPoints; i++) teapot_obj.points[i] *= scale;

  //  allocate memory for vertices on device
  Vec3 *d_points;
  checkCudaErrors(hipMalloc((void **)&d_points, pointsArraySize));
  checkCudaErrors(hipMemcpy(d_points, teapot_obj.points.data(), pointsArraySize, hipMemcpyHostToDevice));

  //  allocate memory for faces on device
  Vec3 *d_faces;
  checkCudaErrors(hipMalloc((void **)&d_faces, facesArraySize));
  checkCudaErrors(hipMemcpy(d_faces, teapot_obj.faces.data(), facesArraySize, hipMemcpyHostToDevice));

  // allocate memory for Hitable list for the triangles of the loaded mesh
  Hitable **d_MeshTriangles;
  checkCudaErrors(hipMalloc((void **)&d_MeshTriangles, nFaces * sizeof(Hitable *)));

  Hitable **d_mesh;
  checkCudaErrors(hipMalloc((void **)&d_mesh, sizeof(Hitable *)));
  // all the prep for the mehs is done, now create the mesh on device from the loaded data
  meshFromTriangleArray<<<1, 1>>>(d_mesh, d_MeshTriangles, d_points, nPoints, d_faces, nFaces);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // free faces and points on device
//  checkCudaErrors(hipFree(d_faces));
//  checkCudaErrors(hipFree(d_points));

  return d_mesh;

}

__global__ void create_mesh_scene(Hitable **mesh, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *d_world = *mesh;
    Vec3 lookfrom(10,10,2);
    Vec3 lookat(0,0,-1);
    float dist_to_focus = (lookfrom-lookat).length();
    float aperture = 0.0;
    *d_camera   = new Camera(lookfrom,
                             lookat,
                             Vec3(0,1,0),
                             40.0,
                             float(nx)/float(ny),
                             aperture,
                             dist_to_focus, 0.0f, 1.0f);
  }

}

__global__ void free_mesh_scene(Hitable **mesh, Hitable **d_world, Camera **d_camera) {
  delete *mesh;
  delete *d_world;
  delete *d_camera;
}

__global__ void create_mesh_and_cornell_box_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, int object_N, Hitable** d_mesh) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    Material *red = new lambertian(new ConstantTexture(Vec3(0.65, 0.05, 0.05)));
    Material *white = new lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73)));
    Material *green = new lambertian(new ConstantTexture(Vec3(0.12, 0.45, 0.15)));
    Material *light = new DiffuseLight(new ConstantTexture(Vec3(15, 15, 15)));
    Material *shiny = new metal(new ConstantTexture(Vec3(1.0, 1.0, 1.0)), 0.03f);

    *(d_list) = new flip_normals(new YZ_Rectangle(0, 555, 0, 555, 555, green));
    *(d_list+1) = new YZ_Rectangle(0, 555, 0, 555, 0, red);
    *(d_list+2) = new XZ_Rectangle(213, 343, 227, 332, 554, light);
    *(d_list+3) = new flip_normals( new XZ_Rectangle(0, 555, 0, 555, 555, white));
    *(d_list+4) = new XZ_Rectangle(0, 555, 0, 555, 0, white);
    *(d_list+5) = new flip_normals( new XY_Rectangle(0, 555, 0, 555, 555, white));

    *(d_list+6) = *d_mesh;




//    *(d_list+6) = new translate( new rotate_y( new Box(Vec3(0, 0, 0), Vec3(165, 165, 165), new dielectric(1.5)), -18), Vec3(130, 0, 65));
//    *(d_list+7) = new translate( new rotate_y( new Box(Vec3(0, 0, 0), Vec3(165, 330, 165), shiny), 15), Vec3(265, 0, 295));



    *d_world = new HitableList(d_list, object_N);
    Vec3 lookfrom(278, 278, -800);
    Vec3 lookat(278, 278, 0);
    float dist_to_focus = 10.0;
    float aperture = 0.0;
    *d_camera = new Camera(lookfrom,
                           lookat,
                           Vec3(0, 1, 0), 40.0,
                           float(nx) / float(ny),
                           aperture,
                           dist_to_focus, 0.0f, 1.0f);
  }
}

__global__ void free_mesh_cornell_scene(Hitable **d_list, Hitable **d_world, Camera **d_camera) {

  //memory leak here for now but using virtual destructors causes crash for some reason..
  for(int i=0; i < 7; i++) {
    delete d_list[i];
  }

  delete *d_world;
  delete *d_camera;
}

hipTextureObject_t createImageTexture(const char *const filename){
  int width, height, channels;
  unsigned char* img = stbi_load(filename, &width, &height, &channels, 0);
  if (!img) {
    // Handle error
    fprintf(stderr, "Failed to load image texture with filename: %s\n", filename);
    exit(2);
  }
//  printf("width: %d, height: %d, channels: %d\n", width, height, channels);

  // allocate texture array on device and copy image data to it

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  hipArray* cuArray;
  hipMallocArray(&cuArray, &channelDesc, width, height);

  hipMemcpy2DToArray(cuArray, 0, 0, img, width * channels, width * channels, height, hipMemcpyHostToDevice);

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeNormalizedFloat;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  stbi_image_free(img);

  return texObj;
}


int main() {

  // read config file

  ConfigParser config("../config.txt");

  const int nx = config.getInt("nx");
  const int ny = config.getInt("ny");
  const int ns = config.getInt("ns");
  const int tx = config.getInt("tx");
  const int ty = config.getInt("ty");
  const int rSEED = config.getInt("rSEED");
  const int object_N = config.getInt("object_N");

  int num_pixels = nx*ny;

  std::cout << "Starting ray tracer: "<< "width: " << nx << ", height: " << ny << ", samples: " << ns << std::endl;

  Hitable **d_mesh = loadMeshFromOBJFile("../models/bunny.obj", 1000.f);

  // load texture into gpu memory
  hipTextureObject_t texObj = createImageTexture("../textures/earthmap1k.png");

  Hitable **d_list;
  checkCudaErrors(   hipMalloc(  (void **)&d_list, object_N*sizeof(Hitable *)));
  Hitable **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hitable *)));
  Camera **d_camera;
  checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));



  create_cornell_box_scene<<<1,1>>>(d_list,d_world, d_camera, nx, ny, object_N);
//  create_spheres_scene<<<1,1>>>(d_list,d_world, d_camera, nx, ny, object_N, texObj);
//  create_mesh_scene<<<1, 1>>>(d_mesh, d_world, d_camera, nx, ny, object_N);
//  create_mesh_and_cornell_box_scene<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, object_N, d_mesh);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // allocate a cuRAND d_rand_state object for every pixel
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));


  dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
  dim3 threads(tx,ty);

  //initialize RNG
  render_init<<<blocks, threads>>>(nx, ny, d_rand_state, rSEED);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // make image to write to
  Image image(nx, ny);

  auto start = std::chrono::high_resolution_clock::now();
  // main render function
  render<<<blocks, threads>>>(image.fb, nx, ny, ns, d_camera, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout << "Time taken: " << duration.count() << " microseconds also known as " << duration.count()/1000000.0 << " seconds" << std::endl;


  image.save("../output.png");
  image.free();

  checkCudaErrors(hipDeviceSynchronize());

//  free_cornell_box_scene<<<1, 1>>>(d_list, d_world, d_camera);
//  free_spheres_scene<<<1, 1>>>(d_list, d_world, d_camera);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_camera));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_rand_state));


  hipDeviceReset();

  return 0;
}